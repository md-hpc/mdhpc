#include "hip/hip_runtime.h"
extern "C" {
#include "pdb_importer.h"
}
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>

#if defined(SIMULATE) && defined(TIME_RUN)
#error Cannot compile with both SIMULATE and TIME_RUN flags
#endif

#if !defined(SIMULATE) && !defined(TIME_RUN)
#error Cannot compile without neither SIMULATE nor TIME_RUN flags
#endif

// minimum max shared memory size per SM across all architectures is 64K
// minimum max resident block per SM across all architectures is 16
// so worst case, each block will have max 4K shared memory

// use profiler to identify optimal size ie. CUDA occupancy API, nvvp
#define MAX_PARTICLES_PER_CELL 128

//#define EPSILON (1.65e-9)                       // ng * m^2 / s^2
#define EPSILON (1.65e11)                       // ng * A^2 / s^2 originally (1.65e-9)
#define ARGON_MASS (39.948 * 1.66054e-15)       // ng
#define SIGMA (3.4f)                            // Angstrom
#define PLUS_1(dimension, length) ((dimension != length - 1) * (dimension + 1))
#define MINUS_1(dimension, length) ((dimension == 0) * length + dimension - 1)
#define GPU_PERROR(err) do {\
    if (err != hipSuccess) {\
        fprintf(stderr,"gpu_perror: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
        exit(err);\
    }\
} while (0);

#define R_MIN (3.13796173693f)                  // r(-4 * V(2^(1/6) * EPSILON))
#define LJMAX_ACCELERATION (6.8829688151e25)    // A(R_MIN) in Angstrom / s^2

__device__ float compute_acceleration(float r_angstrom) {
        if (r_angstrom < R_MIN)
            return LJMAX_ACCELERATION;

        // in A / s^2
        float temp = SIGMA / r_angstrom;
        temp = temp * temp; // ^2
        temp = temp * temp * temp; // ^ 6

        return 24 * EPSILON * (2 * temp * temp - temp) / (r_angstrom * ARGON_MASS);
}

// the meat:
__global__ void force_eval(const struct Cell *cell_list, float *accelerations)
{
    /*
        1D block array will look like this:
                14               14
        | HNNNNNNNNNNNNN | HNNNNNNNNNNNNN | ... | 
            0               14               28    X*Y*Z*14

        Map one block to a home-neighbor tuple (home cell, neighbor cell)
        Map one thread to a particle index in the home cell, which calculates accelerations in 
        a one to all fashion with the particles in the neighbor cell.

        CAREFUL: one of the home-neighbor tuple will actually be a home-home tuple
    */

    // find hcell coordinate based off of block index x
    int home_x = blockIdx.x % CELL_LENGTH_X;
    int home_y = blockIdx.x / CELL_LENGTH_X % CELL_LENGTH_Y;
    int home_z = blockIdx.x / (CELL_LENGTH_Y * CELL_LENGTH_X) % CELL_LENGTH_Z;

    // find ncell coordinate based off of block index y
    int neighbor_x;
    if (blockIdx.y < 9) {
        neighbor_x = PLUS_1(home_x, CELL_LENGTH_X);
    } else {
        neighbor_x = home_x;
    }

    int neighbor_y;
    if (blockIdx.y < 3) {
        neighbor_y = MINUS_1(home_y, CELL_LENGTH_Y);
    } else if (blockIdx.y >= 3 && blockIdx.y <= 5 || blockIdx.y > 11) {
        neighbor_y = home_y;
    } else {
        neighbor_y = PLUS_1(home_y, CELL_LENGTH_Y);
    }

    int neighbor_z;
    if (blockIdx.y % 3 == 0) {
        neighbor_z = PLUS_1(home_z, CELL_LENGTH_Z);
    } else if (blockIdx.y % 3 == 1) {
        neighbor_z = home_z;
    } else {
        neighbor_z = MINUS_1(home_z, CELL_LENGTH_Z);
    }

    int neighbor_idx = neighbor_x + neighbor_y * CELL_LENGTH_X + neighbor_z * CELL_LENGTH_X * CELL_LENGTH_Y;

    // define and assign shared memory
    __shared__ struct Cell neighbor_cell;
    neighbor_cell.particle_ids[threadIdx.x] = cell_list[neighbor_idx].particle_ids[threadIdx.x];
    neighbor_cell.x[threadIdx.x] = cell_list[neighbor_idx].x[threadIdx.x];
    neighbor_cell.y[threadIdx.x] = cell_list[neighbor_idx].y[threadIdx.x];
    neighbor_cell.z[threadIdx.x] = cell_list[neighbor_idx].z[threadIdx.x];

    // for periodic boundary condition
    if (blockIdx.x != neighbor_idx) {
        if (home_x - neighbor_x == CELL_LENGTH_X - 1)
            neighbor_cell.x[threadIdx.x] += (CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST);
        else if (neighbor_x - home_x == CELL_LENGTH_X - 1)
            neighbor_cell.x[threadIdx.x] -= (CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST);
        if (home_y - neighbor_y == CELL_LENGTH_Y - 1)
            neighbor_cell.y[threadIdx.x] += (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST);
        else if (neighbor_y - home_y == CELL_LENGTH_Y - 1)
            neighbor_cell.y[threadIdx.x] -= (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST);
        if (home_z - neighbor_z == CELL_LENGTH_Z - 1)
            neighbor_cell.z[threadIdx.x] += (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST);
        else if (neighbor_z - home_z == CELL_LENGTH_Z - 1)
            neighbor_cell.z[threadIdx.x] -= (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST);
    }

    // synchronizes threads within a block (all threads must complete tasks)
    __syncthreads();

    int reference_id = cell_list[blockIdx.x].particle_ids[threadIdx.x];
    // if particle exists loop through every particle in ncell particle list
    if (reference_id != -1) {
        float reference_x = cell_list[blockIdx.x].x[threadIdx.x];
        float reference_y = cell_list[blockIdx.x].y[threadIdx.x];
        float reference_z = cell_list[blockIdx.x].z[threadIdx.x];

        float reference_ax = 0;
        float reference_ay = 0;
        float reference_az = 0;

        for (int i = 0; i < MAX_PARTICLES_PER_CELL; ++i) {
            int neighbor_id = neighbor_cell.particle_ids[i];
            if (neighbor_id == -1)
                break;

            if (neighbor_idx == blockIdx.x && !(reference_id < neighbor_id))
                continue;

            float diff_x = reference_x - neighbor_cell.x[i];
            float diff_y = reference_y - neighbor_cell.y[i];
            float diff_z = reference_z - neighbor_cell.z[i];

            float norm = sqrtf((diff_x * diff_x) + (diff_y * diff_y) + (diff_z * diff_z));

            float acceleration = compute_acceleration(norm) / norm;
            float ax = acceleration * diff_x;
            float ay = acceleration * diff_y;
            float az = acceleration * diff_z;

            reference_ax += ax;
            reference_ay += ay;
            reference_az += az;

            atomicAdd(&accelerations[neighbor_id * 3], -ax);
            atomicAdd(&accelerations[neighbor_id * 3 + 1], -ay);
            atomicAdd(&accelerations[neighbor_id * 3 + 2], -az);
        }

        atomicAdd(&accelerations[reference_id * 3], reference_ax);
        atomicAdd(&accelerations[reference_id * 3 + 1], reference_ay);
        atomicAdd(&accelerations[reference_id * 3 + 2], reference_az);
    }
}

__global__ void particle_update(struct Cell *cell_list, float *accelerations)
{
    // 1 block -> 1 cell
    // 1 thread -> 1 particle

    int reference_id = cell_list[blockIdx.x].particle_ids[threadIdx.x];
    if (reference_id == -1)
        return;

    cell_list[blockIdx.x].vx[threadIdx.x] += accelerations[reference_id * 3] * TIMESTEP_DURATION_FS;
    cell_list[blockIdx.x].vy[threadIdx.x] += accelerations[reference_id * 3 + 1] * TIMESTEP_DURATION_FS;
    cell_list[blockIdx.x].vz[threadIdx.x] += accelerations[reference_id * 3 + 2] * TIMESTEP_DURATION_FS;

    float x = cell_list[blockIdx.x].x[threadIdx.x] + cell_list[blockIdx.x].vx[threadIdx.x] * TIMESTEP_DURATION_FS;
    x += ((x < 0) - (x > CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST)) * (CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST);
    cell_list[blockIdx.x].x[threadIdx.x] = x;

    float y = cell_list[blockIdx.x].y[threadIdx.x] + cell_list[blockIdx.x].vy[threadIdx.x] * TIMESTEP_DURATION_FS;
    y += ((y < 0) - (y > CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST)) * (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST);
    cell_list[blockIdx.x].y[threadIdx.x] = y;

    float z = cell_list[blockIdx.x].z[threadIdx.x] + cell_list[blockIdx.x].vz[threadIdx.x] * TIMESTEP_DURATION_FS;
    z += ((z < 0) - (z > CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST)) * (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST);
    cell_list[blockIdx.x].z[threadIdx.x] = z;

    accelerations[reference_id] = 0;
}

// update cell lists because particles have moved
__global__ void motion_update(struct Cell *cell_list_src, struct Cell *cell_list_dst)
{
    /*
        1 block per cell
        right now 1 thread per particle in a block
        keeps counter on next free spot on new particle list
        once a -1 in the old particle list is reached, there are no particles to the right
    */
    // get home cell coordinates

    // threadIdx.x is always 0 because we are indexing by blockIdx.x
    int home_x = blockIdx.x % CELL_LENGTH_X;
    int home_y = blockIdx.x / CELL_LENGTH_X % CELL_LENGTH_Y;
    int home_z = blockIdx.x / (CELL_LENGTH_X * CELL_LENGTH_Y) % CELL_LENGTH_Z;

    // location of where thread is in buffer
    __shared__ int free_idx;
    if (threadIdx.x == 0)
        free_idx = 0;
    __syncthreads();

    for (int current_cell_idx = 0; current_cell_idx < CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z; ++current_cell_idx) {
        int current_particle_id = cell_list_src[current_cell_idx].particle_ids[threadIdx.x];
        if (current_particle_id == -1)
            continue;

        float current_particle_x = cell_list_src[current_cell_idx].x[threadIdx.x];
        float current_particle_y = cell_list_src[current_cell_idx].y[threadIdx.x];
        float current_particle_z = cell_list_src[current_cell_idx].z[threadIdx.x];

        int new_cell_x = current_particle_x / CELL_CUTOFF_RADIUS_ANGST;
        int new_cell_y = current_particle_y / CELL_CUTOFF_RADIUS_ANGST;
        int new_cell_z = current_particle_z / CELL_CUTOFF_RADIUS_ANGST;

        if (home_x == new_cell_x && home_y == new_cell_y && home_z == new_cell_z) {
            int idx = atomicAdd(&free_idx, 1);
            cell_list_dst[blockIdx.x].particle_ids[idx] = current_particle_id;
            cell_list_dst[blockIdx.x].x[idx] = current_particle_x;
            cell_list_dst[blockIdx.x].y[idx] = current_particle_y;
            cell_list_dst[blockIdx.x].z[idx] = current_particle_z;
            cell_list_dst[blockIdx.x].vx[idx] = cell_list_src[current_cell_idx].vx[threadIdx.x];
            cell_list_dst[blockIdx.x].vy[idx] = cell_list_src[current_cell_idx].vy[threadIdx.x];
            cell_list_dst[blockIdx.x].vz[idx] = cell_list_src[current_cell_idx].vz[threadIdx.x];
        }
    }

    __syncthreads();

    if (threadIdx.x >= free_idx)
        cell_list_dst[blockIdx.x].particle_ids[threadIdx.x] = -1;

    return;
}


int main(int argc, char **argv) 
{
    if (argc != 3) {
	    printf("Usage: ./cell_list <input_file> <output_file>\n");
	    return 1;
    }

    char *input_file = argv[1];
    char *output_file = argv[2];
    FILE *out = fopen(output_file, "w");
    fprintf(out, "particle_id,x,y,z\n");
    fclose(out);

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE CELL LIST WITH PARTICLE DATA
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // initialize (or import) particle data for simulation
    int particle_count;

    int *host_particle_ids;
    float *host_x;
    float *host_y;
    float *host_z;
    struct Cell host_cell_list[CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z];

    // device_cell_list stores an array of Cells, where each Cell contains a particle_list
    struct Cell *device_cell_list_1;
    struct Cell *device_cell_list_2;
    float *accelerations;

    // import particles from PDB file
    import_atoms(input_file, &host_particle_ids, &host_x, &host_y, &host_z, &particle_count);
    create_cell_list(host_particle_ids, host_x, host_y, host_z, particle_count, host_cell_list, CELL_CUTOFF_RADIUS_ANGST, CELL_LENGTH_X, CELL_LENGTH_Y, CELL_LENGTH_Z);

    // hipMalloc initializes GPU global memory to be used as parameter for GPU kernel
    GPU_PERROR(hipMalloc(&device_cell_list_1, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell)));
    GPU_PERROR(hipMemcpy(device_cell_list_1, host_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyHostToDevice));
    GPU_PERROR(hipMalloc(&device_cell_list_2, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell)));

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE ACCELERATIONS
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /*
        accelerations stores accelerations (in x y z dimensions) of each particle to be used in motion update.
        - index of accelerations is related to particle_id
        - particle_id * 3 gives index of accelerations for x dimension
        - (particle_id * 3) + 1 gives index of y
        - (particle_id * 3) + 2 gives index of y
    */
    GPU_PERROR(hipMalloc(&accelerations, particle_count * 3 * sizeof(float)));
    GPU_PERROR(hipMemset(accelerations, 0, particle_count * 3 * sizeof(float)));

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE PARAMETERS FOR FORCE COMPUTATION AND MOTION UPDATE
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // defines block and thread dimensions
    // dim3 is an integer vector type most commonly used to pass the grid and block dimensions in a kernel invocation [X x Y x Z]
    // there are 2^31 blocks in x dimension while y and z have at most 65536 blocks
    dim3 numBlocksForce(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z, 14);        // (CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * 14) x 1 x 1
    dim3 numBlocksParticle(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z);
    dim3 numBlocksMotion(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z);            // (CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z) x 1 x 1
    dim3 threadsPerBlockForce(MAX_PARTICLES_PER_CELL);                              // MAX_PARTICLES_PER_CELL x 1 x 1
    dim3 threadsPerBlockParticle(MAX_PARTICLES_PER_CELL);                              // MAX_PARTICLES_PER_CELL x 1 x 1
//    dim3 threadsPerBlockMotion(CELL_LENGTH_X, CELL_LENGTH_Y, CELL_LENGTH_Z);  

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // FORCE COMPUTATION AND MOTION UPDATE
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // do force evaluation and motion update for each time step
    // steps are separated to ensure threads are synchronized (that force_eval is done)
    // output of force_eval is stores in device_cell_list and accelerations

#ifdef TIME_RUN
    hipEvent_t time_start;
    hipEvent_t time_stop;
    GPU_PERROR(hipEventCreate(&time_start));
    GPU_PERROR(hipEventCreate(&time_stop));

    GPU_PERROR(hipEventRecord(time_start));
#endif    

    for (int t = 0; t < TIMESTEPS; ++t) {
        if (t % 2 == 0) {
            force_eval<<<numBlocksForce, threadsPerBlockForce>>>(device_cell_list_1, accelerations);
            particle_update<<<numBlocksParticle, threadsPerBlockParticle>>>(device_cell_list_1, accelerations);
            motion_update<<<numBlocksMotion, MAX_PARTICLES_PER_CELL>>>(device_cell_list_1, device_cell_list_2);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_cell_list, device_cell_list_2, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
#endif
        } else {
            force_eval<<<numBlocksForce, threadsPerBlockForce>>>(device_cell_list_2, accelerations);
            particle_update<<<numBlocksParticle, threadsPerBlockParticle>>>(device_cell_list_2, accelerations);
            motion_update<<<numBlocksMotion, MAX_PARTICLES_PER_CELL>>>(device_cell_list_2, device_cell_list_1);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_cell_list, device_cell_list_1, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
#endif
        }
#ifdef SIMULATE
        cell_list_to_csv(host_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z, output_file, "a");
#endif
    }

#ifdef TIME_RUN
    GPU_PERROR(hipEventRecord(time_stop));

    if (TIMESTEPS % 2 == 1) {
        GPU_PERROR(hipMemcpy(host_cell_list, device_cell_list_2, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
    } else {
        GPU_PERROR(hipMemcpy(host_cell_list, device_cell_list_1, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
    }

    GPU_PERROR(hipEventSynchronize(time_stop));
    float elapsed_milliseconds = 0;
    GPU_PERROR(hipEventElapsedTime(&elapsed_milliseconds, time_start, time_stop));
    printf("cell_list_n3l,%d,%f\n", particle_count, elapsed_milliseconds / 1000);

    cell_list_to_csv(host_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z, output_file, "a");
#endif

    GPU_PERROR(hipFree(device_cell_list_1));
    GPU_PERROR(hipFree(device_cell_list_2));
    GPU_PERROR(hipFree(accelerations));

    return 0;
}
