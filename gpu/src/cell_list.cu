#include "hip/hip_runtime.h"
extern "C" {
#include "pdb_importer.h"
}
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>

#if defined(SIMULATE) && defined(TIME_RUN)
#error Cannot compile with both SIMULATE and TIME_RUN flags
#endif

#if !defined(SIMULATE) && !defined(TIME_RUN)
#error Cannot compile without neither SIMULATE nor TIME_RUN flags
#endif

//#define EPSILON (1.65e-9)                       // ng * m^2 / s^2
#define EPSILON (1.65e11)                       // ng * A^2 / s^2 originally (1.65e-9)
#define ARGON_MASS (39.948 * 1.66054e-15)       // ng
#define SIGMA (3.4f)                            // Angstrom
#define PLUS_1(dimension, length) ((dimension != length - 1) * (dimension + 1))
#define MINUS_1(dimension, length) ((dimension == 0) * length + dimension - 1)
#define GPU_PERROR(err) do {\
    if (err != hipSuccess) {\
        fprintf(stderr,"gpu_perror: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
        exit(err);\
    }\
} while (0);

#define R_MIN (3.13796173693f)                  // r(-4 * V(2^(1/6) * EPSILON))
#define LJMAX_ACCELERATION (6.8829688151e25)    // A(R_MIN) in Angstrom / s^2

__device__ float compute_acceleration(float r_angstrom) {
        if (r_angstrom < R_MIN)
            return LJMAX_ACCELERATION;

        // in A / s^2
        float temp = SIGMA / r_angstrom;
        temp = temp * temp; // ^2
        temp = temp * temp * temp; // ^ 6

        return 24 * EPSILON * (2 * temp * temp - temp) / (r_angstrom * ARGON_MASS);
}

__global__ void force_eval(const struct Cell *cell_list, float *accelerations)
{
    int home_x = blockIdx.x % CELL_LENGTH_X;
    int home_y = (blockIdx.x / CELL_LENGTH_X) % CELL_LENGTH_Y;
    int home_z = blockIdx.x / (CELL_LENGTH_Y * CELL_LENGTH_X);

    int neighbor_x;
    switch (blockIdx.y % 3) {
    case 0:
        neighbor_x = MINUS_1(home_x, CELL_LENGTH_X);
        break;
    case 1:
        neighbor_x = home_x;
        break;
    case 2:
        neighbor_x = PLUS_1(home_x, CELL_LENGTH_X);
        break;
    }

    int neighbor_y;
    switch ((blockIdx.y / 3) % 3) {
    case 0:
        neighbor_y = MINUS_1(home_y, CELL_LENGTH_Y);
        break;
    case 1:
        neighbor_y = home_y;
        break;
    case 2:
        neighbor_y = PLUS_1(home_y, CELL_LENGTH_Y);
        break;
    }

    int neighbor_z;
    switch (blockIdx.y / 9) {
    case 0:
        neighbor_z = MINUS_1(home_z, CELL_LENGTH_Z);
        break;
    case 1:
        neighbor_z = home_z;
        break;
    case 2:
        neighbor_z = PLUS_1(home_z, CELL_LENGTH_Z);
        break;
    }

    int neighbor_idx = neighbor_x + neighbor_y * CELL_LENGTH_X + neighbor_z * CELL_LENGTH_X * CELL_LENGTH_Y;

    __shared__ struct Cell neighbor_cell;
    neighbor_cell.particle_ids[threadIdx.x] = cell_list[neighbor_idx].particle_ids[threadIdx.x];
    neighbor_cell.x[threadIdx.x] = cell_list[neighbor_idx].x[threadIdx.x];
    neighbor_cell.y[threadIdx.x] = cell_list[neighbor_idx].y[threadIdx.x];
    neighbor_cell.z[threadIdx.x] = cell_list[neighbor_idx].z[threadIdx.x];

    if (blockIdx.x != neighbor_idx) {
        if (home_x - neighbor_x == CELL_LENGTH_X - 1)
            neighbor_cell.x[threadIdx.x] += (CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST);
        else if (neighbor_x - home_x == CELL_LENGTH_X - 1)
            neighbor_cell.x[threadIdx.x] -= (CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST);
        if (home_y - neighbor_y == CELL_LENGTH_Y - 1)
            neighbor_cell.y[threadIdx.x] += (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST);
        else if (neighbor_y - home_y == CELL_LENGTH_Y - 1)
            neighbor_cell.y[threadIdx.x] -= (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST);
        if (home_z - neighbor_z == CELL_LENGTH_Z - 1)
            neighbor_cell.z[threadIdx.x] += (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST);
        else if (neighbor_z - home_z == CELL_LENGTH_Z - 1)
            neighbor_cell.z[threadIdx.x] -= (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST);
    }

    if (cell_list[blockIdx.x].particle_ids[threadIdx.x] == -1)
        return;

    __syncthreads();

    float reference_x = cell_list[blockIdx.x].x[threadIdx.x];
    float reference_y = cell_list[blockIdx.x].y[threadIdx.x];
    float reference_z = cell_list[blockIdx.x].z[threadIdx.x];

    float reference_ax = 0;
    float reference_ay = 0;
    float reference_az = 0;

    // TODO: consider the situation where home = neighbor
    for (int i = 0; i < MAX_PARTICLES_PER_CELL; ++i) {
        if (neighbor_cell.particle_ids[i] == -1)
            break;

        if (neighbor_cell.particle_ids[i] == cell_list[blockIdx.x].particle_ids[threadIdx.x])
            continue;

        float diff_x = reference_x - neighbor_cell.x[i];
        float diff_y = reference_y - neighbor_cell.y[i];
        float diff_z = reference_z - neighbor_cell.z[i];

        float norm = sqrtf((diff_x * diff_x) + (diff_y * diff_y) + (diff_z * diff_z));

        float acceleration = compute_acceleration(norm) / norm;
        reference_ax += acceleration * diff_x;
        reference_ay += acceleration * diff_y;
        reference_az += acceleration * diff_z;
    }

    int accelerations_block_idx = (blockIdx.x * 27 + blockIdx.y) * MAX_PARTICLES_PER_CELL * 3;
    accelerations[accelerations_block_idx + threadIdx.x] = reference_ax;
    accelerations[accelerations_block_idx + threadIdx.x + MAX_PARTICLES_PER_CELL] = reference_ay;
    accelerations[accelerations_block_idx + threadIdx.x + (MAX_PARTICLES_PER_CELL * 2)] = reference_az;

    return;
}

__global__ void particle_update(struct Cell *cell_list, float *accelerations)
{
    if (cell_list[blockIdx.x].particle_ids[threadIdx.x] == -1)
        return;

    float reference_vx = cell_list[blockIdx.x].vx[threadIdx.x];
    float reference_vy = cell_list[blockIdx.x].vy[threadIdx.x];
    float reference_vz = cell_list[blockIdx.x].vz[threadIdx.x];
    float reference_x = cell_list[blockIdx.x].x[threadIdx.x];
    float reference_y = cell_list[blockIdx.x].y[threadIdx.x];
    float reference_z = cell_list[blockIdx.x].z[threadIdx.x];

    float ax = 0;
    float ay = 0;
    float az = 0;

    for (int i = 0; i < 27; ++i) {
        int accelerations_block_idx = (blockIdx.x * 27 + i) * MAX_PARTICLES_PER_CELL * 3;
        ax += accelerations[accelerations_block_idx + threadIdx.x];
        accelerations[accelerations_block_idx + threadIdx.x] = 0;
        ay += accelerations[accelerations_block_idx + threadIdx.x + MAX_PARTICLES_PER_CELL];
        accelerations[accelerations_block_idx + threadIdx.x + MAX_PARTICLES_PER_CELL] = 0;
        az += accelerations[accelerations_block_idx + threadIdx.x + (MAX_PARTICLES_PER_CELL * 2)];
        accelerations[accelerations_block_idx + threadIdx.x + (MAX_PARTICLES_PER_CELL * 2)] = 0;
    }

    reference_vx += ax * TIMESTEP_DURATION_FS;
    reference_vy += ay * TIMESTEP_DURATION_FS;
    reference_vz += az * TIMESTEP_DURATION_FS;

    float x = reference_x + reference_vx * TIMESTEP_DURATION_FS;
    x += ((x < 0) - (x > CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST)) * (CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST);
    reference_x = x;

    float y = reference_y + reference_vy * TIMESTEP_DURATION_FS;
    y += ((y < 0) - (y > CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST)) * (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST);
    reference_y = y;

    float z = reference_z + reference_vz * TIMESTEP_DURATION_FS;
    z += ((z < 0) - (z > CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST)) * (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST);
    reference_z = z;

    cell_list[blockIdx.x].vx[threadIdx.x] = reference_vx;
    cell_list[blockIdx.x].vy[threadIdx.x] = reference_vy;
    cell_list[blockIdx.x].vz[threadIdx.x] = reference_vz;
    cell_list[blockIdx.x].x[threadIdx.x] = reference_x;
    cell_list[blockIdx.x].y[threadIdx.x] = reference_y;
    cell_list[blockIdx.x].z[threadIdx.x] = reference_z;

    return;
}

__global__ void motion_update(struct Cell *cell_list_src, struct Cell *cell_list_dst)
{
    int home_x = blockIdx.x % CELL_LENGTH_X;
    int home_y = (blockIdx.x / CELL_LENGTH_X) % CELL_LENGTH_Y;
    int home_z = blockIdx.x / (CELL_LENGTH_X * CELL_LENGTH_Y);

    __shared__ int free_idx;
    if (threadIdx.x == 0)
        free_idx = 0;
    __syncthreads();

    // can maybe make the optimization of only looking at neighboring cells on the assumption that particles don't move more than one cell in a timestep
    for (int current_cell_idx = 0; current_cell_idx < CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z; ++current_cell_idx) {
        int current_particle_id = cell_list_src[current_cell_idx].particle_ids[threadIdx.x];
        if (current_particle_id == -1)
            continue;

        float current_particle_x = cell_list_src[current_cell_idx].x[threadIdx.x];
        float current_particle_y = cell_list_src[current_cell_idx].y[threadIdx.x];
        float current_particle_z = cell_list_src[current_cell_idx].z[threadIdx.x];

        int new_cell_x = current_particle_x / CELL_CUTOFF_RADIUS_ANGST;
        int new_cell_y = current_particle_y / CELL_CUTOFF_RADIUS_ANGST;
        int new_cell_z = current_particle_z / CELL_CUTOFF_RADIUS_ANGST;

        if (home_x == new_cell_x && home_y == new_cell_y && home_z == new_cell_z) {
            int idx = atomicAdd(&free_idx, 1);
            cell_list_dst[blockIdx.x].particle_ids[idx] = current_particle_id;
            cell_list_dst[blockIdx.x].x[idx] = current_particle_x;
            cell_list_dst[blockIdx.x].y[idx] = current_particle_y;
            cell_list_dst[blockIdx.x].z[idx] = current_particle_z;
            cell_list_dst[blockIdx.x].vx[idx] = cell_list_src[current_cell_idx].vx[threadIdx.x];
            cell_list_dst[blockIdx.x].vy[idx] = cell_list_src[current_cell_idx].vy[threadIdx.x];
            cell_list_dst[blockIdx.x].vz[idx] = cell_list_src[current_cell_idx].vz[threadIdx.x];
        }
    }

    __syncthreads();

    if (threadIdx.x >= free_idx)
        cell_list_dst[blockIdx.x].particle_ids[threadIdx.x] = -1;

    return;
}

int main(int argc, char **argv)
{
    if (argc != 3) {
	    printf("Usage: ./cell_list <input_file> <output_file>\n");
	    return 1;
    }

    char *input_file = argv[1];
    char *output_file = argv[2];
    FILE *out = fopen(output_file, "w");
    fprintf(out, "particle_id,x,y,z\n");
    fclose(out);

    int particle_count;

    int *host_particle_ids = NULL;
    float *host_x = NULL;
    float *host_y = NULL;
    float *host_z = NULL;
    struct Cell host_cell_list[CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z];

    struct Cell *device_cell_list_1;
    struct Cell *device_cell_list_2;
    float *accelerations;

    import_atoms(input_file, &host_particle_ids, &host_x, &host_y, &host_z, &particle_count);
    create_cell_list(host_particle_ids, host_x, host_y, host_z, particle_count, host_cell_list, CELL_CUTOFF_RADIUS_ANGST, CELL_LENGTH_X, CELL_LENGTH_Y, CELL_LENGTH_Z);

    GPU_PERROR(hipMalloc(&device_cell_list_1, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell)));
    GPU_PERROR(hipMemcpy(device_cell_list_1, host_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyHostToDevice));

    GPU_PERROR(hipMalloc(&device_cell_list_2, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell)));
    GPU_PERROR(hipMalloc(&accelerations, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * 27 * MAX_PARTICLES_PER_CELL * 3 * sizeof(float)));
    GPU_PERROR(hipMemset(accelerations, 0, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * 27 * MAX_PARTICLES_PER_CELL * 3 * sizeof(float)));

    dim3 numBlocksCalculate(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z, 27);
    dim3 numBlocksUpdate(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z);
    dim3 threadsPerBlock(MAX_PARTICLES_PER_CELL);

#ifdef TIME_RUN
    hipEvent_t time_start;
    hipEvent_t time_stop;
    GPU_PERROR(hipEventCreate(&time_start));
    GPU_PERROR(hipEventCreate(&time_stop));

    GPU_PERROR(hipEventRecord(time_start));
#endif

    for (int t = 0; t < TIMESTEPS; ++t) {
        if (t % 2 == 0) {
            force_eval<<<numBlocksCalculate, threadsPerBlock>>>(device_cell_list_1, accelerations);
            particle_update<<<numBlocksUpdate, threadsPerBlock>>>(device_cell_list_1, accelerations);
            motion_update<<<numBlocksUpdate, threadsPerBlock>>>(device_cell_list_1, device_cell_list_2);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_cell_list, device_cell_list_2, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
#endif
        } else {
            force_eval<<<numBlocksCalculate, threadsPerBlock>>>(device_cell_list_2, accelerations);
            particle_update<<<numBlocksUpdate, threadsPerBlock>>>(device_cell_list_2, accelerations);
            motion_update<<<numBlocksUpdate, threadsPerBlock>>>(device_cell_list_2, device_cell_list_1);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_cell_list, device_cell_list_1, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
#endif
        }
#ifdef SIMULATE
        cell_list_to_csv(host_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z, output_file, "a");
#endif
    }

#ifdef TIME_RUN
    GPU_PERROR(hipEventRecord(time_stop));

    if (TIMESTEPS % 2 == 1) {
        GPU_PERROR(hipMemcpy(host_cell_list, device_cell_list_2, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
    } else {
        GPU_PERROR(hipMemcpy(host_cell_list, device_cell_list_1, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
    }

    GPU_PERROR(hipEventSynchronize(time_stop));
    float elapsed_milliseconds = 0;
    GPU_PERROR(hipEventElapsedTime(&elapsed_milliseconds, time_start, time_stop));
    printf("cell_list,%d,%f\n", particle_count, elapsed_milliseconds / 1000);

    cell_list_to_csv(host_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z, output_file, "a");
#endif

    GPU_PERROR(hipFree(device_cell_list_1));
    GPU_PERROR(hipFree(device_cell_list_2));
    GPU_PERROR(hipFree(accelerations));

    return 0;
}
