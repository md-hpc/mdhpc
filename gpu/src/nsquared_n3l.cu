#include "hip/hip_runtime.h"
extern "C" {

#include "pdb_importer.h"

}
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>

#if defined(SIMULATE) && defined(TIME_RUN)
#error Cannot compile with both SIMULATE and TIME_RUN flags
#endif

#if !defined(SIMULATE) && !defined(TIME_RUN)
#error Cannot compile without neither SIMULATE nor TIME_RUN flags
#endif

#define MAX_PARTICLES_PER_BLOCK 32
#define EPSILON (1.65e11)                       // ng * A^2 / s^2 originally (1.65e-9)
#define ARGON_MASS (39.948 * 1.66054e-15)       // ng
#define SIGMA (3.4f)                            // Angstrom
#define GPU_PERROR(err) do {\
    if (err != hipSuccess) {\
        fprintf(stderr,"gpu_perror: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
        exit(err);\
    }\
} while (0);

#define R_MIN (3.13796173693f)                  // r(-4 * V(2^(1/6) * EPSILON))
#define LJMAX_ACCELERATION (6.8829688151e25)    // A(R_MIN) in Angstrom / s^2

__device__ float compute_acceleration(float r_angstrom) {
        if (r_angstrom < R_MIN)
            return LJMAX_ACCELERATION;

        // in A / s^2
        float temp = SIGMA / r_angstrom;
        temp = temp * temp; // ^2
        temp = temp * temp * temp; // ^ 6

        return 24 * EPSILON * (2 * temp * temp - temp) / (r_angstrom * ARGON_MASS);
}

__global__ void calculate_accelerations(float *particle_id, float *src_x, float *src_y, float *src_z,
                                        float *vx, float *vy, float *vz, float *accelerations, int particle_count)
{
    __shared__ float shared_id[MAX_PARTICLES_PER_BLOCK];
    __shared__ float shared_x[MAX_PARTICLES_PER_BLOCK];
    __shared__ float shared_y[MAX_PARTICLES_PER_BLOCK];
    __shared__ float shared_z[MAX_PARTICLES_PER_BLOCK];

    // each thread gets a particle as a reference particle
    int reference_particle_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // extra threads can exit 
    if (reference_particle_idx >= particle_count)
        return; 

    // calculate acceleration block index depending on block index
    const int accelerations_block_idx = blockIdx.x * particle_count * 3;

    float reference_particle_id = particle_id[reference_particle_idx];
    float reference_x = src_x[reference_particle_idx]; 
    float reference_y = src_y[reference_particle_idx]; 
    float reference_z = src_z[reference_particle_idx]; 

    float ax = 0;
    float ay = 0;
    float az = 0;

    // accumulate accelerations for every other particle
    for (int i = 0; i < particle_count; i += MAX_PARTICLES_PER_BLOCK) {
        shared_id[threadIdx.x] = particle_id[i + threadIdx.x];
        shared_x[threadIdx.x] = src_x[i + threadIdx.x];
        shared_y[threadIdx.x] = src_y[i + threadIdx.x];
        shared_z[threadIdx.x] = src_z[i + threadIdx.x];
        //__syncthreads();

        for (int j = 0; j < MAX_PARTICLES_PER_BLOCK; ++j) {
            int jj = (threadIdx.x + j) % MAX_PARTICLES_PER_BLOCK;
            if (shared_id[jj] == reference_particle_id)
                continue;

            float neighbor_x = shared_x[jj];
            float neighbor_y = shared_y[jj];
            float neighbor_z = shared_z[jj];

            float diff_x = reference_x - neighbor_x;
            float diff_y = reference_y - neighbor_y;
            float diff_z = reference_z - neighbor_z;
            float norm = sqrtf((diff_x * diff_x) + (diff_y * diff_y) + (diff_z * diff_z));
            
            float acceleration = compute_acceleration(norm);
            float dax = acceleration * diff_x / norm;
            float day = acceleration * diff_y / norm;
            float daz = acceleration * diff_z / norm;
            ax += dax;
            ay += day;
            az += daz;

            if (reference_particle_id < shared_id[jj]) {
                accelerations[accelerations_block_idx + (i + jj) * 3] -= dax;
                accelerations[accelerations_block_idx + (i + jj) * 3 + 1] -= day;
                accelerations[accelerations_block_idx + (i + jj) * 3 + 2] -= daz;
            }
        }
        //__syncthreads();
    }

    accelerations[accelerations_block_idx + threadIdx.x * 3] = ax;
    accelerations[accelerations_block_idx + threadIdx.x * 3 + 1] = ay;
    accelerations[accelerations_block_idx + threadIdx.x * 3 + 2] = az;
}

__global__ void position_update(float *src_x, float *src_y, float *src_z,
                                float *vx, float *vy, float *vz,
                                float *dst_x, float *dst_y, float *dst_z,
                                int particle_count, float *accelerations)
{
    int reference_particle_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (reference_particle_idx >= particle_count)
        return;

    float ax = 0;
    float ay = 0;
    float az = 0;

    const int accelerations_block_size = particle_count * sizeof(float) * 3;
    for (int i = 0; i < gridDim.x; ++i) {
        ax += accelerations[reference_particle_idx + accelerations_block_size * i];
        ay += accelerations[reference_particle_idx + accelerations_block_size * i + 1];
        az += accelerations[reference_particle_idx + accelerations_block_size * i + 2];
    }

    // calculate velocity for reference particle
    float reference_x = src_x[reference_particle_idx];
    float reference_y = src_y[reference_particle_idx];
    float reference_z = src_z[reference_particle_idx];
    float reference_vx = vx[reference_particle_idx]; 
    float reference_vy = vy[reference_particle_idx]; 
    float reference_vz = vz[reference_particle_idx]; 
    reference_vx += ax * TIMESTEP_DURATION_FS;
    reference_vy += ay * TIMESTEP_DURATION_FS;
    reference_vz += az * TIMESTEP_DURATION_FS;

    // get new reference particle position taking into account periodic boundary conditions
    float x = reference_x + reference_vx * TIMESTEP_DURATION_FS;
    x += ((x < 0) - (x > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_x = x;
 
    float y = reference_y + reference_vy * TIMESTEP_DURATION_FS;
    y += ((y < 0) - (y > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_y = y;

    float z = reference_z + reference_vz * TIMESTEP_DURATION_FS;
    z += ((z < 0) - (z > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_z = z;

    vx[reference_particle_idx] = reference_vx;
    vy[reference_particle_idx] = reference_vy;
    vz[reference_particle_idx] = reference_vz;
    dst_x[reference_particle_idx] = reference_x;
    dst_y[reference_particle_idx] = reference_y;
    dst_z[reference_particle_idx] = reference_z;
}

int main(int argc, char **argv) 
{
    if (argc != 3) {
        printf("Usage: ./nsquared_n3l <input_file> <output_file>\n");
        return 1; 
    }
    
    char *input_file = argv[1];
    char *output_file = argv[2];
    FILE *out = fopen(output_file, "w");
    fprintf(out, "particle_id,x,y,z\n");

    int particle_count;

    int *host_particle_ids = NULL;
    float *host_x = NULL;
    float *host_y = NULL;
    float *host_z = NULL;

    float *device_particle_ids;
    float *device_x_1;
    float *device_y_1;
    float *device_z_1;
    float *device_x_2;
    float *device_y_2;
    float *device_z_2;
    float *vx;
    float *vy;
    float *vz;
    float *accelerations;

    import_atoms(input_file, &host_particle_ids, &host_x, &host_y, &host_z, &particle_count);

    GPU_PERROR(hipMalloc(&device_particle_ids, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_x_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_y_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_z_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_x_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_y_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_z_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vx, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vy, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vz, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&accelerations, ((particle_count - 1) / MAX_PARTICLES_PER_BLOCK + 1) * particle_count * sizeof(float) * 3));

    GPU_PERROR(hipMemcpy(device_particle_ids, host_particle_ids, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_x_1, host_x, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_y_1, host_y, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_z_1, host_z, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemset(vx, 0, particle_count * sizeof(float)));
    GPU_PERROR(hipMemset(vy, 0, particle_count * sizeof(float)));
    GPU_PERROR(hipMemset(vz, 0, particle_count * sizeof(float)));

    // set parameters
    dim3 numBlocks((particle_count - 1) / MAX_PARTICLES_PER_BLOCK + 1);
    dim3 threadsPerBlock(MAX_PARTICLES_PER_BLOCK);

#ifdef TIME_RUN
    hipEvent_t time_start;
    hipEvent_t time_stop;
    GPU_PERROR(hipEventCreate(&time_start));
    GPU_PERROR(hipEventCreate(&time_stop));

    GPU_PERROR(hipEventRecord(time_start));
#endif

    for (int t = 0; t < TIMESTEPS; ++t) {
        GPU_PERROR(hipMemset(accelerations, 0, ((particle_count - 1) / MAX_PARTICLES_PER_BLOCK + 1) * particle_count * sizeof(float) * 3));
        if (t % 2 == 0) {
            calculate_accelerations<<<numBlocks, threadsPerBlock>>>(device_particle_ids, device_x_1, device_y_1, device_z_1, device_x_2, device_y_2, device_z_2, accelerations, particle_count);
            position_update<<<numBlocks, threadsPerBlock>>>(device_x_1, device_y_1, device_z_1, vx, vy, vz, device_x_2, device_y_2, device_z_2, particle_count, accelerations);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_x, device_x_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_y, device_y_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_z, device_z_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
        } else {
            calculate_accelerations<<<numBlocks, threadsPerBlock>>>(device_particle_ids, device_x_2, device_y_2, device_z_2, device_x_1, device_y_1, device_z_1, accelerations, particle_count);
            position_update<<<numBlocks, threadsPerBlock>>>(device_x_2, device_y_2, device_z_2, vx, vy, vz, device_x_1, device_y_1, device_z_1, particle_count, accelerations);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_x, device_x_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_y, device_y_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_z, device_z_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
        }
#ifdef SIMULATE
        for (int i = 0; i < particle_count; ++i) {
            fprintf(out, "%d,%f,%f,%f\n", host_particle_ids[i], host_x[i], host_y[i], host_z[i]);
        }
        fprintf(out, "\n");
#endif
    }

#ifdef TIME_RUN
    GPU_PERROR(hipEventRecord(time_stop));

    if (TIMESTEPS % 2 == 1) {
        GPU_PERROR(hipMemcpy(host_x, device_x_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_y, device_y_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_z, device_z_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
    } else {
        GPU_PERROR(hipMemcpy(host_x, device_x_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_y, device_y_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_z, device_z_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
    }

    GPU_PERROR(hipEventSynchronize(time_stop));
    float elapsed_milliseconds = 0;
    GPU_PERROR(hipEventElapsedTime(&elapsed_milliseconds, time_start, time_stop));
    printf("nsquared_n3l,%d,%f\n", particle_count, elapsed_milliseconds / 1000);
        
    for (int i = 0; i < particle_count; ++i) {
        fprintf(out, "%d,%f,%f,%f\n", host_particle_ids[i], host_x[i], host_y[i], host_z[i]);
    }
#endif

    GPU_PERROR(hipFree(device_particle_ids));
    GPU_PERROR(hipFree(device_x_1));
    GPU_PERROR(hipFree(device_y_1));
    GPU_PERROR(hipFree(device_z_1));
    GPU_PERROR(hipFree(device_x_2));
    GPU_PERROR(hipFree(device_y_2));
    GPU_PERROR(hipFree(device_z_2));
    GPU_PERROR(hipFree(vx));
    GPU_PERROR(hipFree(vy));
    GPU_PERROR(hipFree(vz));

    return 0;
}
