#include "hip/hip_runtime.h"
extern "C" {

#include "pdb_importer.h"

}
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>

#if defined(SIMULATE) && defined(TIME_RUN)
#error Cannot compile with both SIMULATE and TIME_RUN flags
#endif

#if !defined(SIMULATE) && !defined(TIME_RUN)
#error Cannot compile without neither SIMULATE nor TIME_RUN flags
#endif

#define MAX_PARTICLES_PER_BLOCK 1024
#define EPSILON (1.65e11)                       // ng * A^2 / s^2 originally (1.65e-9)
#define ARGON_MASS (39.948 * 1.66054e-15)       // ng
#define SIGMA (3.4f)                            // Angstrom
#define GPU_PERROR(err) do {\
    if (err != hipSuccess) {\
        fprintf(stderr,"gpu_perror: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
        exit(err);\
    }\
} while (0);

#define R_MIN (3.13796173693f)                  // r(-4 * V(2^(1/6) * EPSILON))
#define LJMAX_ACCELERATION (6.8829688151e25)    // A(R_MIN) in Angstrom / s^2

__device__ float compute_acceleration(float r_angstrom) {
        if (r_angstrom < R_MIN)
            return LJMAX_ACCELERATION;

        // in A / s^2
        float temp = SIGMA / r_angstrom;
        temp = temp * temp; // ^2
        temp = temp * temp * temp; // ^ 6

        return 24 * EPSILON * (2 * temp * temp - temp) / (r_angstrom * ARGON_MASS);
}

__global__ void timestep(float *particle_id, float *src_x, float *src_y, float *src_z,
                         float *vx, float *vy, float *vz, float *dst_x, float *dst_y,
                         float *dst_z, int particle_count)
{
    // initialize shared memory - shared between threads in each block
    __shared__ float shared_id[MAX_PARTICLES_PER_BLOCK];
    __shared__ float shared_x[MAX_PARTICLES_PER_BLOCK];
    __shared__ float shared_y[MAX_PARTICLES_PER_BLOCK];
    __shared__ float shared_z[MAX_PARTICLES_PER_BLOCK];

    // each thread gets a particle as a reference particle
    int reference_particle_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // extra threads can exit
    if (reference_particle_idx >= particle_count)
        return; 

    // get reference particle positions and id
    float reference_particle_id = particle_id[reference_particle_idx];
    float reference_x = src_x[reference_particle_idx]; 
    float reference_y = src_y[reference_particle_idx]; 
    float reference_z = src_z[reference_particle_idx]; 

    // accumulate accelerations for every other particle
    float ax = 0;
    float ay = 0;
    float az = 0;
    for (int i = 0; i < particle_count; i += MAX_PARTICLES_PER_BLOCK) {
        shared_id[threadIdx.x] = particle_id[i + threadIdx.x];
        shared_x[threadIdx.x] = src_x[i + threadIdx.x];
        shared_y[threadIdx.x] = src_y[i + threadIdx.x];
        shared_z[threadIdx.x] = src_z[i + threadIdx.x];

#if MAX_PARTICLES_PER_BLOCK > 32
        __syncthreads();
#endif

        for (int j = 0; j < MAX_PARTICLES_PER_BLOCK; ++j) {
            // dont calculate acceleration with itself
            if (shared_id[(threadIdx.x + j) % MAX_PARTICLES_PER_BLOCK] == reference_particle_id)
                continue;
            // get neighbor particle positions from shared memory
            float neighbor_x = shared_x[(threadIdx.x + j) % MAX_PARTICLES_PER_BLOCK];
            float neighbor_y = shared_y[(threadIdx.x + j) % MAX_PARTICLES_PER_BLOCK];
            float neighbor_z = shared_z[(threadIdx.x + j) % MAX_PARTICLES_PER_BLOCK];
            // use temp variables to optimize
            float diff_x = reference_x - neighbor_x;
            float diff_y = reference_y - neighbor_y;
            float diff_z = reference_z - neighbor_z;
            // get norm for acceleration calculation
            float norm = sqrtf((diff_x * diff_x) + (diff_y * diff_y) + (diff_z * diff_z));

            // compute scalar acceleration and apply to xyz directions 
            float acceleration = compute_acceleration(norm);
            ax += acceleration * (reference_x - neighbor_x) / norm;
            ay += acceleration * (reference_y - neighbor_y) / norm;
            az += acceleration * (reference_z - neighbor_z) / norm;
        }
#if MAX_PARTICLES_PER_BLOCK > 32
        __syncthreads();
#endif
    }

    // calculate velocity for reference particle
    float reference_vx = vx[reference_particle_idx]; 
    float reference_vy = vy[reference_particle_idx]; 
    float reference_vz = vz[reference_particle_idx]; 
    reference_vx += ax * TIMESTEP_DURATION_FS;
    reference_vy += ay * TIMESTEP_DURATION_FS;
    reference_vz += az * TIMESTEP_DURATION_FS;

    // get new reference particle position taking into account periodic boundary conditions
    float x = reference_x + reference_vx * TIMESTEP_DURATION_FS;
    x += ((x < 0) - (x > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_x = x;
 
    float y = reference_y + reference_vy * TIMESTEP_DURATION_FS;
    y += ((y < 0) - (y > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_y = y;

    float z = reference_z + reference_vz * TIMESTEP_DURATION_FS;
    z += ((z < 0) - (z > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_z = z;

    // write velocity and positions of particle back to global memory
    vx[reference_particle_idx] = reference_vx;
    vy[reference_particle_idx] = reference_vy;
    vz[reference_particle_idx] = reference_vz;
    dst_x[reference_particle_idx] = reference_x;
    dst_y[reference_particle_idx] = reference_y;
    dst_z[reference_particle_idx] = reference_z;
}

int main(int argc, char **argv) 
{
    if (argc != 3) {
        printf("Usage: ./nsquared_shared <input_file> <output_file>\n");
        return 1; 
    }
    
    char *input_file = argv[1];
    char *output_file = argv[2];
    FILE *out = fopen(output_file, "w");
    fprintf(out, "particle_id,x,y,z\n");

    int particle_count;

    int *host_particle_ids = NULL;
    float *host_x = NULL;
    float *host_y = NULL;
    float *host_z = NULL;

    float *device_particle_ids;
    float *device_x_1;
    float *device_y_1;
    float *device_z_1;
    float *device_x_2;
    float *device_y_2;
    float *device_z_2;
    float *vx;
    float *vy;
    float *vz;

    import_atoms(input_file, &host_particle_ids, &host_x, &host_y, &host_z, &particle_count);

    GPU_PERROR(hipMalloc(&device_particle_ids, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_x_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_y_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_z_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_x_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_y_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_z_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vx, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vy, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vz, particle_count * sizeof(float)));

    GPU_PERROR(hipMemcpy(device_particle_ids, host_particle_ids, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_x_1, host_x, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_y_1, host_y, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_z_1, host_z, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemset(vx, 0, particle_count * sizeof(float)));
    GPU_PERROR(hipMemset(vy, 0, particle_count * sizeof(float)));
    GPU_PERROR(hipMemset(vz, 0, particle_count * sizeof(float)));

    // set parameters
    dim3 numBlocks((particle_count - 1) / MAX_PARTICLES_PER_BLOCK + 1);
    dim3 threadsPerBlock(MAX_PARTICLES_PER_BLOCK);

#ifdef TIME_RUN
    hipEvent_t time_start;
    hipEvent_t time_stop;
    GPU_PERROR(hipEventCreate(&time_start));
    GPU_PERROR(hipEventCreate(&time_stop));

    GPU_PERROR(hipEventRecord(time_start));
#endif

    for (int t = 0; t < TIMESTEPS; ++t) {
        if (t % 2 == 0) {
            timestep<<<numBlocks, threadsPerBlock>>>(device_particle_ids, device_x_1, device_y_1, device_z_1, vx, vy, vz, device_x_2, device_y_2, device_z_2, particle_count);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_x, device_x_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_y, device_y_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_z, device_z_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
        } else {
            timestep<<<numBlocks, threadsPerBlock>>>(device_particle_ids, device_x_2, device_y_2, device_z_2, vx, vy, vz, device_x_1, device_y_1, device_z_1, particle_count);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_x, device_x_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_y, device_y_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_z, device_z_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
        }
#ifdef SIMULATE
        for (int i = 0; i < particle_count; ++i) {
            fprintf(out, "%d,%f,%f,%f\n", host_particle_ids[i], host_x[i], host_y[i], host_z[i]);
        }
        fprintf(out, "\n");
#endif
    }

#ifdef TIME_RUN
    GPU_PERROR(hipEventRecord(time_stop));

    if (TIMESTEPS % 2 == 1) {
        GPU_PERROR(hipMemcpy(host_x, device_x_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_y, device_y_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_z, device_z_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
    } else {
        GPU_PERROR(hipMemcpy(host_x, device_x_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_y, device_y_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_z, device_z_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
    }

    GPU_PERROR(hipEventSynchronize(time_stop));
    float elapsed_milliseconds = 0;
    GPU_PERROR(hipEventElapsedTime(&elapsed_milliseconds, time_start, time_stop));
    printf("nsquared_shared,%d,%f\n", particle_count, elapsed_milliseconds / 1000);
        
    for (int i = 0; i < particle_count; ++i) {
        fprintf(out, "%d,%f,%f,%f\n", host_particle_ids[i], host_x[i], host_y[i], host_z[i]);
    }
#endif

    GPU_PERROR(hipFree(device_particle_ids));
    GPU_PERROR(hipFree(device_x_1));
    GPU_PERROR(hipFree(device_y_1));
    GPU_PERROR(hipFree(device_z_1));
    GPU_PERROR(hipFree(device_x_2));
    GPU_PERROR(hipFree(device_y_2));
    GPU_PERROR(hipFree(device_z_2));
    GPU_PERROR(hipFree(vx));
    GPU_PERROR(hipFree(vy));
    GPU_PERROR(hipFree(vz));

    return 0;
}
