#include "hip/hip_runtime.h"
extern "C" {

#include "pdb_importer.h"

}
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <math.h>

#if defined(SIMULATE) && defined(TIME_RUN)
#error Cannot compile with both SIMULATE and TIME_RUN flags
#endif

#if !defined(SIMULATE) && !defined(TIME_RUN)
#error Cannot compile without neither SIMULATE nor TIME_RUN flags
#endif

#define MAX_PARTICLES_PER_BLOCK 1024
#define EPSILON (1.65e11)                       // ng * A^2 / s^2 originally (1.65e-9)
#define ARGON_MASS (39.948 * 1.66054e-15)       // ng
#define SIGMA (3.4f)                            // Angstrom
#define GPU_PERROR(err) do {\
    if (err != hipSuccess) {\
        fprintf(stderr,"gpu_perror: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
        exit(err);\
    }\
} while (0);

#define R_MIN (3.13796173693f)                  // r(-4 * V(2^(1/6) * EPSILON))
#define LJMAX_ACCELERATION (6.8829688151e25)    // A(R_MIN) in Angstrom / s^2

#ifdef VALIDATE
__device__ float compute_potential(float r_angstrom) {
        // validation code for getting potential energy
        float temp = SIGMA / r_angstrom;
        temp = temp * temp; // ^2
        temp = temp * temp * temp; // ^ 6
        float potential = 4 * EPSILON * ((temp * temp) - temp);
        return potential;
}
#endif

__device__ float compute_acceleration(float r_angstrom) {
        if (r_angstrom < R_MIN)
            return LJMAX_ACCELERATION;

        // in A / s^2
        float temp = SIGMA / r_angstrom;
        temp = temp * temp; // ^2
        temp = temp * temp * temp; // ^ 6

        return 24 * EPSILON * (2 * temp * temp - temp) / (r_angstrom * ARGON_MASS);
}

__global__ void timestep(float *particle_id, float *src_x, float *src_y, float *src_z,
                         float *vx, float *vy, float *vz, float *dst_x, float *dst_y,
                         float *dst_z, int particle_count, float *device_pe)
{
    // each thread gets a particle as a reference particle
    int reference_particle_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // extra threads can exit 
    if (reference_particle_idx >= particle_count)
        return; 

    // get reference particle positions
    float reference_x = src_x[reference_particle_idx]; 
    float reference_y = src_y[reference_particle_idx]; 
    float reference_z = src_z[reference_particle_idx]; 

    // accumulate accelerations for every other particle (i == 1)
    float ax = 0;
    float ay = 0;
    float az = 0;

    for (int i = 1; i < particle_count; ++i) {
        // use temp variables to optimize
        float diff_x = reference_x - src_x[(reference_particle_idx + i) % particle_count];
        float diff_y = reference_y - src_y[(reference_particle_idx + i) % particle_count];
        float diff_z = reference_z - src_z[(reference_particle_idx + i) % particle_count];

        // get new particle position differences taking into account periodic boundary conditions
        diff_x += ((diff_x < -UNIVERSE_LENGTH / 2) - (diff_x > UNIVERSE_LENGTH / 2)) * UNIVERSE_LENGTH;
        diff_y += ((diff_y < -UNIVERSE_LENGTH / 2) - (diff_y > UNIVERSE_LENGTH / 2)) * UNIVERSE_LENGTH;
        diff_z += ((diff_z < -UNIVERSE_LENGTH / 2) - (diff_z > UNIVERSE_LENGTH / 2)) * UNIVERSE_LENGTH;

        // get norm for acceleration calculation
        float norm = sqrtf((diff_x * diff_x) + (diff_y * diff_y) + (diff_z * diff_z));

        // compute scalar acceleration and apply to xyz directions 
        float acceleration = compute_acceleration(norm) / norm;
        ax += acceleration * diff_x;
        ay += acceleration * diff_y;
        az += acceleration * diff_z;
#ifdef VALIDATE
        device_pe[i] = compute_potential(norm);
#endif
    }

    // obtain current velocity of reference particle
    float reference_vx = vx[reference_particle_idx]; 
    float reference_vy = vy[reference_particle_idx]; 
    float reference_vz = vz[reference_particle_idx]; 
    // calculate velocity for reference particle
    reference_vx += ax * TIMESTEP_DURATION_FS;
    reference_vy += ay * TIMESTEP_DURATION_FS;
    reference_vz += az * TIMESTEP_DURATION_FS;

    // get new reference particle position taking into account periodic boundary conditions
    float x = reference_x + reference_vx * TIMESTEP_DURATION_FS;
    x += ((x < 0) - (x > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_x = x;
 
    float y = reference_y + reference_vy * TIMESTEP_DURATION_FS;
    y += ((y < 0) - (y > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_y = y;

    float z = reference_z + reference_vz * TIMESTEP_DURATION_FS;
    z += ((z < 0) - (z > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_z = z;

    // write velocity and positions of particle back to global memory
    vx[reference_particle_idx] = reference_vx;
    vy[reference_particle_idx] = reference_vy;
    vz[reference_particle_idx] = reference_vz;
    dst_x[reference_particle_idx] = reference_x;
    dst_y[reference_particle_idx] = reference_y;
    dst_z[reference_particle_idx] = reference_z;
}

int main(int argc, char **argv) 
{
    if (argc != 3) {
        printf("Usage: ./nsquared <input_file> <output_file>\n");
        return 1; 
    }
    
    char *input_file = argv[1];
    char *output_file = argv[2];
    FILE *out = fopen(output_file, "w");
    fprintf(out, "particle_id,x,y,z\n");

    int particle_count;

    int *host_particle_ids = NULL;
    float *host_x = NULL;
    float *host_y = NULL;
    float *host_z = NULL;

    float *device_particle_ids;
    float *device_x_1;
    float *device_y_1;
    float *device_z_1;
    float *device_x_2;
    float *device_y_2;
    float *device_z_2;
    float *vx;
    float *vy;
    float *vz;
    float *device_pe;

    import_atoms(input_file, &host_particle_ids, &host_x, &host_y, &host_z, &particle_count);
    float *host_vx = (float *)malloc(particle_count * sizeof(float));
    float *host_vy = (float *)malloc(particle_count * sizeof(float));
    float *host_vz = (float *)malloc(particle_count * sizeof(float));
    float *host_pe = (float *)malloc(particle_count * sizeof(float));

    GPU_PERROR(hipMalloc(&device_particle_ids, particle_count * sizeof(int)));
    GPU_PERROR(hipMalloc(&device_x_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_y_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_z_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_x_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_y_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_z_2, particle_count * sizeof(float)));
    // need for validation
    GPU_PERROR(hipMalloc(&vx, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vy, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vz, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_pe, particle_count * sizeof(float)));

    GPU_PERROR(hipMemcpy(device_particle_ids, host_particle_ids, particle_count * sizeof(int), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_x_1, host_x, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_y_1, host_y, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_z_1, host_z, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemset(vx, 0.0f, particle_count * sizeof(float)));
    GPU_PERROR(hipMemset(vy, 0.0f, particle_count * sizeof(float)));
    GPU_PERROR(hipMemset(vz, 0.0f, particle_count * sizeof(float)));

    // set parameters
    dim3 numBlocks((particle_count - 1) / MAX_PARTICLES_PER_BLOCK + 1);
    dim3 threadsPerBlock(MAX_PARTICLES_PER_BLOCK);

#ifdef TIME_RUN
    hipEvent_t time_start;
    hipEvent_t time_stop;
    GPU_PERROR(hipEventCreate(&time_start));
    GPU_PERROR(hipEventCreate(&time_stop));

    GPU_PERROR(hipEventRecord(time_start));
#endif

#ifdef VALIDATE
    float momentum_x = 0;
    float momentum_y = 0;
    float momentum_z = 0; 

    float potential_energy = 0;
    float kinetic_energy = 0;
#endif

    for (int t = 0; t < TIMESTEPS; ++t) {
        if (t % 2 == 0) {
            timestep<<<numBlocks, threadsPerBlock>>>(device_particle_ids, device_x_1, device_y_1, device_z_1, vx, vy, vz, device_x_2, device_y_2, device_z_2, particle_count, device_pe);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_x, device_x_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_y, device_y_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_z, device_z_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
#ifdef VALIDATE
            GPU_PERROR(hipMemcpy(host_pe, device_pe, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_vx, vx, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_vy, vy, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_vz, vz, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
        } else {
            timestep<<<numBlocks, threadsPerBlock>>>(device_particle_ids, device_x_2, device_y_2, device_z_2, vx, vy, vz, device_x_1, device_y_1, device_z_1, particle_count, device_pe);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_x, device_x_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_y, device_y_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_z, device_z_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
#ifdef VALIDATE
            GPU_PERROR(hipMemcpy(host_pe, device_pe, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_vx, vx, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_vy, vy, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_vz, vz, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
        }
#ifdef SIMULATE
        for (int i = 0; i < particle_count; ++i) {
            fprintf(out, "%d,%f,%f,%f\n", host_particle_ids[i], host_x[i], host_y[i], host_z[i]);
        }
        fprintf(out, "\n");
#endif
#ifdef VALIDATE
        for (int i = 0; i < particle_count; ++i) {
            // accumulate momentum
            momentum_x += host_vx[i];
            momentum_y += host_vy[i];
            momentum_z += host_vz[i];
            // calculate kinetic energy through velocity
            float particle_kinetic_energy = 0.5 * ARGON_MASS * ((host_vx[i] * host_vx[i]) + (host_vy[i] * host_vy[i]) + (host_vz[i] * host_vz[i]));
            potential_energy += host_pe[i];
            kinetic_energy += particle_kinetic_energy;
        }
        printf("potential %.12f + kinetic %.12f = total energy %.12f\n", potential_energy, kinetic_energy, potential_energy + kinetic_energy);
        printf("momentum x: %.12f\n", momentum_x);
        printf("momentum y: %.12f\n", momentum_y);
        printf("momentum z: %.12f\n", momentum_z);
#endif
    }

#ifdef TIME_RUN
    GPU_PERROR(hipEventRecord(time_stop));

    if (TIMESTEPS % 2 == 1) {
        GPU_PERROR(hipMemcpy(host_x, device_x_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_y, device_y_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_z, device_z_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
    } else {
        GPU_PERROR(hipMemcpy(host_x, device_x_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_y, device_y_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_z, device_z_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
    }

    GPU_PERROR(hipEventSynchronize(time_stop));
    float elapsed_milliseconds = 0;
    GPU_PERROR(hipEventElapsedTime(&elapsed_milliseconds, time_start, time_stop));
    printf("nsquared,%d,%f\n", particle_count, elapsed_milliseconds / 1000);

    for (int i = 0; i < particle_count; ++i) {
        fprintf(out, "%d,%f,%f,%f\n", host_particle_ids[i], host_x[i], host_y[i], host_z[i]);
    }
#endif

    GPU_PERROR(hipFree(device_particle_ids));
    GPU_PERROR(hipFree(device_x_1));
    GPU_PERROR(hipFree(device_y_1));
    GPU_PERROR(hipFree(device_z_1));
    GPU_PERROR(hipFree(device_x_2));
    GPU_PERROR(hipFree(device_y_2));
    GPU_PERROR(hipFree(device_z_2));
    GPU_PERROR(hipFree(vx));
    GPU_PERROR(hipFree(vy));
    GPU_PERROR(hipFree(vz));

    return 0;
}
